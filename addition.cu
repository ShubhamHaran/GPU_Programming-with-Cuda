#include <iostream>
#include <math.h>
#include "hip/hip_runtime.h"

// Kernel function to add the elements of two arrays
__global__
void add(int n, float* x, float* y)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1 << 20;
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    float* x, * y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 2.0f;
        y[i] = 1.0f;
    }
    int b=8,t=1024;

    // Run kernel on 1M elements on the GPU
    add <<<b, t>>> (N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    std::cout<<"Time for the GPU kernel of blocksize "<<b<< " thread size "<<t<<" number of elements "<<N<<": "<<time<<" ms"<<std::endl;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i = 0; i < N; i++) {
        x[i] = 2.0f;
        y[i] = 1.0f;
    }
    b=1;t=1;

    // Run kernel on 1M elements on the GPU
    add <<<b, t>>> (N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    std::cout<<"Time for the CPU kernel: "<<time<<" ms"<<std::endl;
    

    

    // Free memory
    hipFree(x);
    hipFree(y);
    return 0;
}
